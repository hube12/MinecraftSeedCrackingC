#ifndef CUDA_CRACKER_RANDOM
#define CUDA_CRACKER_RANDOM
#include <stdio.h>
#include "hip/hip_runtime.h"

unsigned long long __device__ initSeed(unsigned long long seed)
{
    return (seed ^ (unsigned long long) 0x5deece66d) & ((1LLU << 48u) - 1);
}

long signed __device__ nextBits(unsigned long long *seed, unsigned int bits) {
	unsigned long long newSeed = 0;
    if (bits < 1) { bits = 1; }
    else if (bits > 32) { bits = 32; }
    newSeed = (*seed * 0x5deece66d + 0xb);
    newSeed &= ((1LLU << 48u) - 1);
    *seed = newSeed;
    return (long) (newSeed >> (48u - bits));
}

int __device__ nextInt(unsigned long long *seed, long unsigned bound)
{
    if (!(bound & (bound - 1))) { // bound is a power of 2
        return (long signed) ((bound * (unsigned long long) nextBits(seed, 31)) >> 31u);
    }
    else {
		long signed bits = nextBits(seed, 31);
		long signed val = bits % bound;
		while ((bits - val + (long long signed)bound - 1) < 0) { // Apparently nvcc needs the explicit casting here
			bits = nextBits(seed, 31);
			val = bits % bound;
		}
		return val;
	}
}

float __device__ nextFloat(unsigned long long *seed) {
	return nextBits(seed, 24) / (float) (1llu << 24u);
}

unsigned long long __device__ structure_seed(unsigned long seed, unsigned int pillar_seed) {
    unsigned long long currentSeed;
    currentSeed = (seed << 16u & (unsigned long long) 0xFFFF00000000) | (pillar_seed << 16u) |
                  (seed & (unsigned long long) 0xFFFF);
    currentSeed = ((currentSeed - 0xb) * 0xdfe05bcb1365) & (unsigned long long) 0xffffffffffff;
    currentSeed = ((currentSeed - 0xb) * 0xdfe05bcb1365) & (unsigned long long) 0xffffffffffff;
    currentSeed ^= 0x5DEECE66Du;
    return currentSeed;
}
#endif
