#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <chrono>
#include "CudaRandom.cu"
#include "CudaStructure.cu"
#include "hip/hip_runtime.h"

#define MAX_CANDIDATES 16
#define NUM_BLOCKS 1024
#define BLOCK_SIZE 1024
#define SEEDS_PER_BLOCK (1LLU << 32u)/(NUM_BLOCKS*BLOCK_SIZE)

__device__ unsigned long long d_seed_candidates[MAX_CANDIDATES];
__device__ unsigned long long d_candidate_count = 0;

__device__ void add_candidate(unsigned long long seed)
{
    unsigned long long index = atomicAdd(&d_candidate_count, 1); // return old value, perfect for array index
    if (index<MAX_CANDIDATES) {
        d_seed_candidates[index] = seed;
        printf("Found candidate %llu: %llu\n", index+1, seed);
    }
}

__global__ void searchKernel(unsigned int pillar_seed, struct StructureInfo *array, int num_structs, int max_fails)
{
    unsigned long long startSeed = (blockIdx.x*blockDim.x+threadIdx.x)*SEEDS_PER_BLOCK;
    unsigned long long currentSeed;
    int matches;
    for (unsigned long long offset=0; offset<SEEDS_PER_BLOCK; offset++) {
        currentSeed = structure_seed(startSeed+offset, pillar_seed);
        matches = validate_seed(currentSeed, array, num_structs, max_fails);
        if (matches==num_structs) {
            add_candidate(currentSeed);
        }
        else if (matches>3){
            printf("Good Seed: %llu matches %d Structures\n", currentSeed, matches);
        }
    }
}


void cuda_handler(unsigned int pillar_seed, const std::vector<Structure> &arrayStruct)
{
    int max_fails = 1; // up to one structure can be wrong

    int num_structs = arrayStruct.size();
    size_t array_size_bytes = num_structs*sizeof(struct StructureInfo);
    struct StructureInfo *array = (struct StructureInfo *)malloc(array_size_bytes);
    struct StructureInfo *d_array;
    unsigned long long candidate_count;
    unsigned long long *seed_candidates;
    std::ofstream file;

    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

    // Convert to normal c structs
    structure_vector_to_array(arrayStruct, array);

    // Copy to GPU
    hipMalloc((void **)&d_array, array_size_bytes);
    hipMemcpy(d_array, array, array_size_bytes, hipMemcpyHostToDevice);

    // run search
    searchKernel<<<NUM_BLOCKS,BLOCK_SIZE>>>(pillar_seed, d_array, num_structs, max_fails);
    hipDeviceSynchronize();



    file.open("log_process0", std::ios::out | std::ios::trunc);
    if (file.is_open()) {
        // Retrieve candidate count
        hipMemcpyFromSymbol(&candidate_count, HIP_SYMBOL(d_candidate_count), sizeof(unsigned long long));
        printf("Found %llu candidates\n", candidate_count);
        if (candidate_count) {
            if (candidate_count>MAX_CANDIDATES) {
                candidate_count = MAX_CANDIDATES;
                printf("GPU only stored the first %llu\n", MAX_CANDIDATES);
            }
            // Retrieve candidates
            seed_candidates = (unsigned long long *) malloc(candidate_count*sizeof(unsigned long long));
            hipMemcpyFromSymbol(seed_candidates, HIP_SYMBOL(d_seed_candidates), candidate_count*sizeof(unsigned long long));
            for (int i=0; i<candidate_count; i++) {
                file << seed_candidates[i] << std::endl;
            }
        }
        hipDeviceReset();
        file.close();
    } else {
        throw std::runtime_error("log file was not loaded");
    }

    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);
    std::cout << "It took me " << time_span.count() << " seconds"<< std::endl;

    exit(0);
}

std::vector<unsigned long long> cuda_assemble_logs() {
    std::vector<unsigned long long> partials_possible_seed;
    std::remove("final_log.txt");
    std::ofstream log("final_log.txt", std::ios_base::out | std::ios::app);
    if (log.is_open()) {
        std::ifstream partial_log("log_process0", std::ios_base::in);
        if (partial_log.is_open()) {
            std::string line;
            while (std::getline(partial_log, line)) {
                log<<line<<std::endl;
                partials_possible_seed.push_back(std::stoull(line));
            }
            partial_log.close();
            std::remove("log_process0");
            bool failed = !std::ifstream("log_process0");
            if (!failed) { std::perror("Error deleting file"); std::cout<<"Error deleting file"<<std::endl;}

        } else {
            throw std::runtime_error("log file was not loaded");
        }
        log.close();
    } else {
        throw std::runtime_error("Log main file was not loaded");
    }
    std::cout<<"Log file was recompiled"<<std::endl;
    return partials_possible_seed;
}

