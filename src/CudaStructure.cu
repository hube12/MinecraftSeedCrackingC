#include "Parser.hpp"
#include "CudaRandom.cu"
#include "hip/hip_runtime.h"

// always positive modulo, ie range 0 to <mod-1>
#define MOD(val,mod) ((mod+(val%mod))%mod)

struct StructureInfo {
	long long chunkX;
	long long chunkZ;
	long long incompleteRand;
	int modulus;
	int nextIntBound; // 0: check float, >0: 2x randInt(bound), <0: 2x randInt(-bound)
};

void structure_vector_to_array(std::vector<Structure> arrayStruct, struct StructureInfo *array) {
	int i = 0;
	for (Structure el :arrayStruct) {
		array[i].chunkX	= el.chunkX;
		array[i].chunkZ	= el.chunkZ;
		array[i].incompleteRand = el.incompleteRand;
		array[i].modulus = el.modulus;

		switch (el.typeStruct) {
			case 's': //old structures: igloo, witch hut, desert temple, jungle temple, village
				array[i].nextIntBound=-24;
				break;
			case 'w': //shipwreck
				array[i].nextIntBound=-8;
				break;
			case 'e': //end cities
				array[i].nextIntBound=9;
				break;
			case 'o': //ocean monuments
				array[i].nextIntBound=27;
				break;
			case 'm': //mansions
				array[i].nextIntBound=60;
				break;
			case 'r': //ruins
				array[i].nextIntBound=8;
				break;
			case 't': //treasures
				array[i].nextIntBound=0; // special case since it uses float
				break;
		}
		i++;
	}
}

int __device__ validate_seed(unsigned long long seed, struct StructureInfo *structs, int num_structs, int max_fails) {
	struct StructureInfo *s;
	unsigned long long workingSeed;
	int mod, bound, k, m, matches=0, fails=0;
	for (int sid=0; sid<num_structs; sid++) {
		s = structs+sid;
		bound = s->nextIntBound;
		workingSeed = initSeed(seed + s->incompleteRand);
		if (!bound) { // treasure
			if (nextFloat(&workingSeed)<0.01) {
				matches++;
				continue;
			}
		}
		else{
			if (bound<0) { // shipwreck and old structures (igloo, witch hut, desert temple, jungle temple, village)
				k = nextInt(&workingSeed, -bound);
				m = nextInt(&workingSeed, -bound);
			}
			else { // new structures: end city, ocean monument, mansion, ruin
				k = (nextInt(&workingSeed, bound) + nextInt(&workingSeed, bound)) / 2;
				m = (nextInt(&workingSeed, bound) + nextInt(&workingSeed, bound)) / 2;
			}
			mod = s->modulus;
			if (k==MOD(s->chunkX, mod) && m==MOD(s->chunkZ, mod)) {
				matches++;
				continue;
			}
			fails++;
			if (fails>=max_fails) {
				return matches;
			}
		}
	}
	// success
	return matches;
}
